#include <malloc.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <fstream>
#include "headers.h"

int main(void) {
	FILE *out1, *out2, *inp, *begin, *end;
	inp = fopen("inp.dat", "r");
	out1 = fopen("particles.dat", "w");
	out2 = fopen("param.dat", "w");
	begin = fopen("begin.dat", "w");
	end = fopen("end.dat", "w");
	double dt, p0, lattice_constant,
	epsilon, sigma, T = 0, kb=1,
	kin_E, pot_E, total_E, V, R, Lx,
	max_valuex, max_valuey, max_valuez;
	int n, nz, iter;
	int t0 = time(0);
	vector *h_ParticleX, *d_ParticleX;
	vector *h_ParticleP, *d_ParticleP;
	vector *h_ParticleA, *d_ParticleA;
	vector *d_init_particleX;
	vector *h_init_particleP, *d_init_particleP;
	double *h_R, *d_R;
	double *h_V, *d_V;
	double *h_kin_E, *d_kin_E;
	double *h_pot_E, *d_pot_E;

	//input
	fscanf(inp, "%d", &n);
	fscanf(inp, "%d", &nz);
	fscanf(inp, "%d", &iter);
	fscanf(inp, "%lf", &dt);
	fscanf(inp, "%lf", &Lx);
	fscanf(inp, "%lf", &p0);
	fscanf(inp, "%lf", &epsilon);
	fscanf(inp, "%lf", &sigma);
	fclose(inp);

	size_t vsize = nz * n * n * sizeof(vector);
	size_t size = nz * n * n * sizeof(double);

	// host memory allocation
	h_ParticleX = (vector *) malloc(vsize);
	h_ParticleP = (vector *) malloc(vsize);
	h_ParticleA = (vector *) malloc(vsize);
	h_init_particleP = (vector *) malloc(vsize);
	h_R = (double *) malloc(size);
	h_V = (double *) malloc(size);
	h_kin_E = (double *) malloc(size);
	h_pot_E = (double *) malloc(size);

	// device memory allocation
    hipMalloc(&d_ParticleX, vsize);
	hipMalloc(&d_ParticleP, vsize);
	hipMalloc(&d_ParticleA, vsize);
	hipMalloc(&d_init_particleX, vsize);
	hipMalloc(&d_init_particleP, vsize);
	hipMalloc(&d_R, size);
	hipMalloc(&d_V, size);
	hipMalloc(&d_kin_E, size);
	hipMalloc(&d_pot_E, size);

	srand(time(NULL));

	//initialization of threads geometry
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE_Z);
    double GridSize = (n * n) / BLOCK_SIZE;
    double GridSizeZ = nz / BLOCK_SIZE_Z;
    dim3 dimGrid(GridSize, GridSize, GridSizeZ);

	//system size initialization
	lattice_constant = Lx / (n - 1);
	max_valuex = lattice_constant * n;
	max_valuey = max_valuex;
	max_valuez = lattice_constant * nz;

	Init_coordinates<<< dimGrid, dimBlock>>>(d_ParticleX,
			d_init_particleX, n, nz, lattice_constant);

	Init_impulse(h_ParticleP, h_init_particleP, n, nz, p0);

	hipMemcpy(d_ParticleP, h_ParticleP, vsize, hipMemcpyHostToDevice);
	hipMemcpy(d_init_particleP, h_init_particleP, vsize, hipMemcpyHostToDevice);

	Init_acceleration<<< dimGrid, dimBlock>>>(d_ParticleX, d_ParticleA, n, nz,
			lattice_constant, max_valuex, max_valuey, max_valuez, epsilon, sigma);

	//main part
	for (int time_count = 0; time_count < iter; time_count++){

	//calculation of energy and mean functions
		if (time_count % 10 == 0){
			Energy_and_mean_functions<<< dimGrid, dimBlock>>>(d_ParticleX,
					d_ParticleP, d_init_particleX, d_init_particleP, d_kin_E,
					d_pot_E, d_R, d_V, n, nz, lattice_constant, max_valuex,
					max_valuey, max_valuez, epsilon, sigma);

			hipMemcpy(h_ParticleX, d_ParticleX, vsize, hipMemcpyDeviceToHost);
			hipMemcpy(h_ParticleP, d_ParticleP, vsize, hipMemcpyDeviceToHost);
			hipMemcpy(h_ParticleA, d_ParticleA, vsize, hipMemcpyDeviceToHost);
			hipMemcpy(h_kin_E, d_kin_E, size, hipMemcpyDeviceToHost);
			hipMemcpy(h_pot_E, d_pot_E, size, hipMemcpyDeviceToHost);
			hipMemcpy(h_R, d_R, size, hipMemcpyDeviceToHost);
			hipMemcpy(h_V, d_V, size, hipMemcpyDeviceToHost);

			pot_E = 0;
			kin_E = 0;
			V = 0;
			R = 0;
			for(int i = 0; i < nz * n * n; i++){
				kin_E += h_kin_E[i];
				pot_E += h_pot_E[i];
				R += h_R[i];
				V += h_V[i];
			}
			total_E = kin_E + pot_E * 0.5;
			T += kin_E / nz / n / n / kb;
			V = V / (n * n * nz);
			R = R / (n * n * nz);

		//output
			for(int i = 0; i < nz * n * n; i++){
				fprintf(out1,"%10.10lf   %10.10lf  %10.10lf  %10.10lf "
						"  %10.10lf   %10.10lf  %10.10lf  %10.10lf  %10.10lf\n",
					h_ParticleX[i].x, h_ParticleX[i].y, h_ParticleX[i].z,
					h_ParticleP[i].x, h_ParticleP[i].y, h_ParticleP[i].z,
					h_ParticleA[i].x, h_ParticleA[i].y, h_ParticleA[i].z);
			}
			fprintf(out1,"\n \n");
			fprintf(out2,"%d %10.10lf %10.10lf %10.10lf %10.10lf %10.10lf %10.10lf\n",
					time_count, kin_E, pot_E*0.5, total_E, T/time_count, V, R);

			if (time_count == 0){
				for(int i = 0; i < n * n * nz; i++){
					fprintf(begin,"%10.10lf   %10.10lf  %10.10lf\n",
							h_ParticleX[i].x, h_ParticleX[i].y, h_ParticleX[i].z);
				}
			}
			if (time_count == iter - 10){
				for(int i = 0; i < n * n * nz; i++){
					fprintf(end,"%10.10lf   %10.10lf  %10.10lf\n",
							h_ParticleX[i].x, h_ParticleX[i].y, h_ParticleX[i].z);
				}
			}
		}

		New_coordinates<<< dimGrid, dimBlock>>>(d_ParticleX, d_ParticleP, d_ParticleA,
				n, nz, lattice_constant, dt, max_valuex, max_valuey, max_valuez);

		New_impulse_and_acceleration<<< dimGrid, dimBlock>>>(d_ParticleX, d_ParticleP,
				d_ParticleA, n, nz, lattice_constant, max_valuex, max_valuey, max_valuez,
				epsilon, dt, sigma);
	}
	printf("time %d s\n ", time(0) - t0);
	fclose(out1);
	fclose(out2);
	fclose(begin);
	fclose(end);
	hipFree(d_ParticleA);
	hipFree(d_ParticleX);
	hipFree(d_ParticleP);
	hipFree(d_kin_E);
	hipFree(d_pot_E);
	hipFree(d_R);
	hipFree(d_V);
}
